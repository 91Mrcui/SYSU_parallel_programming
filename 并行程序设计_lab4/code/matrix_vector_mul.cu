#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "read_data.h"

__global__ void matrixVectorMul(float* A, float* b, float* c, int rows, int cols) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rows) {
        float sum = 0.0f;
        for (int j = 0; j < cols; j++) {
            sum += A[tid * cols + j] * b[j];
        }
        c[tid] = sum;
    }
}

void matrixVectorMultiplication(std::vector<float>& A, std::vector<float>& b, std::vector<float>& c, int rows, int cols) {
    // Device memory allocation
    float *d_A, *d_b, *d_c;
    hipMalloc((void**)&d_A, rows * cols * sizeof(float));
    hipMalloc((void**)&d_b, cols * sizeof(float));
    hipMalloc((void**)&d_c, rows * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;
    matrixVectorMul<<<gridSize, blockSize>>>(d_A, d_b, d_c, rows, cols);

    // Copy result from device to host
    hipMemcpy(c.data(), d_c, rows * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    //修改这里的路径，读入二进制文件和输出二进制文件
    string read_dir="data/test5.in";
    string save_dir="output/res5.out";
    std::vector<float> A ;
    std::vector<std::vector<float> >array_2d; 
    std::vector<float> b ;

    read(A,array_2d,b,read_dir);
    //generate_data(A,array_2d,b,4096);

    int rows = array_2d.size();
    int cols = array_2d[0].size();

    std::vector<float> c(rows);

    clock_t start, finish;
    start = clock();
    matrixVectorMultiplication(A, b, c, rows, cols);
    finish=clock();

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < rows; i++) {
        printf("%.5f ",c[i]);
    }
    std::cout << std::endl;
    // Print using time
    std::cout <<"using time: "<<1000*double(finish - start) / CLOCKS_PER_SEC<<" ms"<<std::endl;

    write(c,save_dir);
    return 0;
}
