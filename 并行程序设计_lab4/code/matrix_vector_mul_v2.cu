#include "hip/hip_runtime.h"
//v2版本，用了常量内存储存向量b

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "read_data.h"

__constant__ float d_b[2048];

__global__ void matrixVectorMul(float* A,float* c, int rows, int cols) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rows) {
        float sum = 0.0f;
        for (int j = 0; j < cols; j++) {
            sum += A[tid * cols + j] * d_b[j];
        }
        c[tid] = sum;
    }
}


void matrixVectorMultiplication(const std::vector<float>& A, const std::vector<float>& b, std::vector<float>& c, int rows, int cols) {
    // Device memory allocation
    float *d_A, *d_c;
    hipMalloc((void**)&d_A, rows * cols * sizeof(float));
    hipMalloc((void**)&d_c, rows * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_b), b.data(), cols * sizeof(float));

    // Launch kernel
    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;
    matrixVectorMul<<<gridSize, blockSize>>>(d_A, d_c, rows, cols);

    // Copy result from device to host
    hipMemcpy(c.data(), d_c, rows * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_c);
}


int main() {
    string read_dir="data/test1.in";
    string save_dir="output/res1.out";
    std::vector<float> A ;
    std::vector<std::vector<float> >array_2d; 
    std::vector<float> b ;
    read(A,array_2d,b,read_dir);
    //generate_data(A,array_2d,b,2048);

    int rows = array_2d.size();
    int cols = array_2d[0].size();

    std::vector<float> c(rows);

    clock_t start, finish;
    start = clock();
    matrixVectorMultiplication(A, b, c, rows, cols);
    finish=clock();

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < rows; i++) {
        printf("%.5f ",c[i]);
    }
    std::cout << std::endl;
    // Print using time
    std::cout <<"using time: "<<1000*double(finish - start) / CLOCKS_PER_SEC<<" ms"<<std::endl;
    write(c,save_dir);
    return 0;
}
