#include "hip/hip_runtime.h"
//v3版本，用了纹理内存进行优化，将内核函数中的内存访问从全局内存改为从纹理内存中读取数据

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "read_data.h"

texture<float, 1, hipReadModeElementType> texA;
texture<float, 1, hipReadModeElementType> texB;

__global__ void matrixVectorMul(float* c, int rows, int cols) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rows) {
        float sum = 0.0f;
        for (int j = 0; j < cols; j++) {
            sum += tex1Dfetch(texA, tid * cols + j) * tex1Dfetch(texB, j);
        }
        c[tid] = sum;
    }
}

void matrixVectorMultiplication(std::vector<float>& A, std::vector<float>& b, std::vector<float>& c, int rows, int cols) {
    // Device memory allocation
    float *d_A, *d_b, *d_c;
    hipMalloc((void**)&d_A, rows * cols * sizeof(float));
    hipMalloc((void**)&d_b, cols * sizeof(float));
    hipMalloc((void**)&d_c, rows * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), cols * sizeof(float), hipMemcpyHostToDevice);

    // Bind texture memory
    hipBindTexture(NULL, texA, d_A, rows * cols * sizeof(float));
    hipBindTexture(NULL, texB, d_b, cols * sizeof(float));

    // Launch kernel
    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;
    matrixVectorMul<<<gridSize, blockSize>>>(d_c, rows, cols);

    // Copy result from device to host
    hipMemcpy(c.data(), d_c, rows * sizeof(float), hipMemcpyDeviceToHost);

    // Unbind texture memory
    hipUnbindTexture(texA);
    hipUnbindTexture(texB);

    // Free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    string read_dir="data/test1.in";
    string save_dir="output/res1.out";
    std::vector<float> A ;
    std::vector<std::vector<float> >array_2d; 
    std::vector<float> b ;

    read(A,array_2d,b,read_dir);
    //generate_data(A,array_2d,b,4096);
    int rows = array_2d.size();
    int cols = array_2d[0].size();

    std::vector<float> c(rows);

    clock_t start, finish;
    start = clock();
    matrixVectorMultiplication(A, b, c, rows, cols);
    finish=clock();

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < rows; i++) {
        //std::cout << c[i] << " ";
        printf("%.5f ",c[i]);
    }
    std::cout << std::endl;
    // Print using time
    std::cout <<"using time: "<<1000*double(finish - start) / CLOCKS_PER_SEC<<" ms"<<std::endl;
    write(c,save_dir);
    return 0;
}
