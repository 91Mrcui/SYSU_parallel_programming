#include "hip/hip_runtime.h"
//v4版本，用了cublas进行优化
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipblas.h>
#include "read_data.h"

void matrixVectorMultiplication(std::vector<float>& A, std::vector<float>& b, std::vector<float>& c, int rows, int cols) {
    // Device memory allocation
    float *d_A, *d_b, *d_c;
    hipMalloc((void**)&d_A, rows * cols * sizeof(float));
    hipMalloc((void**)&d_b, cols * sizeof(float));
    hipMalloc((void**)&d_c, rows * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), cols * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS initialization
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Matrix-vector multiplication using cuBLAS
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemv(handle, HIPBLAS_OP_T, cols, rows, &alpha, d_A, cols, d_b, 1, &beta, d_c, 1);

    // Copy result from device to host
    hipMemcpy(c.data(), d_c, rows * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_c);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);
}

int main() {
    string read_dir="data/test1.in";
    string save_dir="output/res1.out";
    std::vector<float> A;
    std::vector<std::vector<float>> array_2d;
    std::vector<float> b;

    read(A, array_2d, b, read_dir);

    int rows = array_2d.size();
    int cols = array_2d[0].size();

    std::vector<float> c(rows);

    clock_t start, finish;
    start = clock();
    matrixVectorMultiplication(A, b, c, rows, cols);
    finish = clock();

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < rows; i++) {
        printf("%.5f ", c[i]);
    }
    std::cout << std::endl;

    // Print execution time
    std::cout << "Using time: " << 1000 * double(finish - start) / CLOCKS_PER_SEC << " ms" << std::endl;
    write(c,save_dir);
    return 0;
}
