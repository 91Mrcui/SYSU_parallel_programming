#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#define BLOCK_SIZE 16
#include "deal_binary.h"
using std::string;

// CUDA核函数，计算以每个元素为中心的窗口中的熵
// 计算每个元素的熵的核函数（使用共享内存优化）
__global__ void calculateEntropy_share(int* input, float* output, int width, int height) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // 定义共享内存
    __shared__ int shared_input[8 + 4][8 + 4];
    // 计算线程在共享内存中的索引
    int shared_row = threadIdx.x + 2;
    int shared_col = threadIdx.y + 2;
    // 将数据从全局内存复制到共享内存
    if (row < height && col < width) {
        int global_index = col * height + row;
        shared_input[shared_row][shared_col] = input[global_index];
    }
    // 线程同步，确保数据复制完成
    __syncthreads();
    if (row < height && col < width) {
        float entropy = 0;
        // 记录每个数字出现过的次数
        int record[16];
        // 窗口内元素总数
        int count = 0, x, y;
        for (int i = 0; i < 16; i++) {
            record[i] = 0;
        }
        for (int i = -2; i <= 2; i++) {
            for (int j = -2; j <= 2; j++) {
                x = shared_col + i;
                y = shared_row + j;
                printf("%d %d\n",x,y);
                int value = shared_input[y][x];
                record[value]++;
                count++;
            }
        }
        // 计算熵值
        for (int i = 0; i < 16; i++) {
            //entropy -= (float)record[i] * (log_table[record[i]]-log_table[count]) / count;
            float prob = (float)(record[i]) / count;
            //printf("%f\n",prob);
            if (prob > 0.0f)
                entropy -= prob * log2f(prob);
        }
        output[col * height + row] = entropy;
    }
}

// CUDA核函数，计算以每个元素为中心的窗口中的熵
__global__ void calculateEntropy(int* input, float* output, int row,int col)
{
    int c = blockIdx.y * blockDim.y + threadIdx.y;
    int r = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < row && c < col)
    {
        int windowSize = 5;
        int windowStartRow = r - 2;
        int windowStartCol = c - 2;
        int windowEndRow = windowStartRow + 4;
        int windowEndCol = windowStartCol + 4;

       // 边界处理
        if (windowStartRow < 0)
            windowStartRow = 0;
        if (windowStartCol < 0)
            windowStartCol = 0;
        if (windowEndRow >= row)
            windowEndRow = row - 1;
        if (windowEndCol >= col)
            windowEndCol = col - 1;

        float entropy = 0.0f;
        int windowElements = (windowEndRow - windowStartRow + 1) * (windowEndCol - windowStartCol + 1);

        // 计算窗口内元素的频率
        int frequency[16] = { 0 };
        for (int i = windowStartRow; i <= windowEndRow; i++)
        {
            for (int j = windowStartCol; j <= windowEndCol; j++)
            {
                int value = input[i * col + j];
                frequency[value]++;
            }
        }

        // 计算熵
        for (int k = 0; k < 16; k++)
        {
            float prob = static_cast<float>(frequency[k]) / windowElements;
            if (prob > 0.0f)
                entropy -= prob * log2f(prob);
        }

        output[r * col + c] = entropy;
    }
}

int main()
{
    int sizes[] = { 5, 16, 128, 1024, 2048, 3000 ,4000};
    srand(static_cast<unsigned int>(1234));
    std::vector<std::vector<int> > array;
    std::vector<float> res;
    read(array,res,"test/test.in","test/test.out");
    int row=array.size();
    int col=array[0].size();


    int* hostInput = new int[row*col];
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            hostInput[i*col+j]=array[i][j];
        }
    }

    int* deviceInput;
    hipMalloc((void**)&deviceInput, row * col * sizeof(int));
    hipMemcpy(deviceInput, hostInput, row * col * sizeof(int), hipMemcpyHostToDevice);

    float* hostOutput = new float[row * col];
    float* deviceOutput;
    hipMalloc((void**)&deviceOutput, row * col * sizeof(float));
    // 定义CUDA的网格和块大小
    // 设置grid、block
    dim3 block_size(8, 8);
    dim3 grid_size((col + block_size.x - 1) / block_size.x, (row + block_size.y - 1) / block_size.y);
    dim3 gridSize((row + BLOCK_SIZE - 1) / BLOCK_SIZE, (col + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    calculateEntropy<<<gridSize, blockSize>>>(deviceInput, deviceOutput, row,col);
    //calculateEntropy<< <grid_size, block_size >> >(deviceInput, deviceOutput, row,col);
    hipMemcpy(hostOutput, deviceOutput, row* col * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<std::vector<float> > entropyArray;
    entropyArray.resize(row);
    for(int i=0;i<row;i++){
        entropyArray[i].resize(col);
    }
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            array[i][j]=hostInput[i*col+j];
            entropyArray[i][j]=hostOutput[i*col+j];
        }
    }

    write(array,entropyArray,"output/share_mem.in","output/share_mem.out");
    
    // 释放内存
    delete[] hostInput;
    delete[] hostOutput;
    hipFree(deviceInput);
    hipFree(deviceOutput);


    return 0;
}
