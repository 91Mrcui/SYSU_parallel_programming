#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#define BLOCK_SIZE 16
#include "deal_binary.h"
using std::string;

// CUDA核函数，计算以每个元素为中心的窗口中的熵
__global__ void calculateEntropy(int* input, float* output, int row,int col)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("hhh\n");
    double logTable[26];
    for (int k = 1; k <= 25; k++)
    {
        logTable[k] = log2f(k);
        //printf("%f\n",logTable[k-1]);
    }
    //printf("kkk\n");
    if (index < row * col)
    {
        int r = index / col;
        int c = index % col;

        int windowSize = 5;
        int windowStartRow = r - 2;
        int windowStartCol = c - 2;
        int windowEndRow = windowStartRow + 4;
        int windowEndCol = windowStartCol + 4;

        // 边界处理
        if (windowStartRow < 0)
            windowStartRow = 0;
        if (windowStartCol < 0)
            windowStartCol = 0;
        if (windowEndRow >= row)
            windowEndRow = row - 1;
        if (windowEndCol >= col)
            windowEndCol = col - 1;

        float entropy = 0.0f;
        int windowElements = (windowEndRow - windowStartRow + 1) * (windowEndCol - windowStartCol + 1);

        // 计算窗口内元素的频率
        int frequency[16] = { 0 };
        for (int i = windowStartRow; i <= windowEndRow; i++)
        {
            for (int j = windowStartCol; j <= windowEndCol; j++)
            {
                int value = input[i * col + j];
                frequency[value]++;
            }
        }

        // 计算熵
        for (int k = 0; k < 16; k++)
        {
            float prob = static_cast<float>(frequency[k]) / windowElements;
            //printf("%d \n",frequency[k]);
            if (prob > 0.0f){
                if(frequency[k] >= 1 && frequency[k] <= 25)
                entropy -= prob * (logTable[frequency[k]]-logTable[windowElements]);
                else 
                entropy -= prob * log2f(prob);
            }
        }

        output[index] = entropy;
    }
}

int main()
{
    int sizes[] = { 5, 16, 128, 1024, 2048, 3000, 4000 };
    srand(static_cast<unsigned int>(1234));
    /*
    for (int i = 0; i < 7; i++)
    {
        int size = sizes[i];
        
        // 随机生成二维数组
        int* hostInput = new int[size * size];
        for (int j = 0; j < size * size; j++)
            hostInput[j] = rand() % 16;

        int* deviceInput;
        hipMalloc((void**)&deviceInput, size * size * sizeof(int));
        hipMemcpy(deviceInput, hostInput, size * size * sizeof(int), hipMemcpyHostToDevice);

        float* hostOutput = new float[size * size];
        float* deviceOutput;
        hipMalloc((void**)&deviceOutput, size * size * sizeof(float));

        // 定义CUDA的网格和块大小
        int gridSize = (size * size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        int blockSize = BLOCK_SIZE;

        // 预计算对数表
        float logTable[25];
        for (int k = 0; k < 25; k++)
        {
            logTable[k] = log2f(static_cast<float>(k + 1));
            //printf("%f ",logTable[k]);
        }
        //printf("\n");

        clock_t start, finish;
        start = clock();
        // 调用CUDA核函数
        calculateEntropy<<<gridSize, blockSize>>>(deviceInput, deviceOutput, size, logTable);

        hipMemcpy(hostOutput, deviceOutput, size * size * sizeof(float), hipMemcpyDeviceToHost);
        finish = clock();
        // 输出结果
        std::cout << "Array Size: " << size << " using time: " << 1000 * double(finish - start) / CLOCKS_PER_SEC << " ms" << std::endl;

        // 释放内存
        delete[] hostInput;
        delete[] hostOutput;
        hipFree(deviceInput);
        hipFree(deviceOutput);
    }
    */
    std::vector<std::vector<int> > array;
    std::vector<float> res;
    read(array,res,"test/test.in","test/test.out");
    int row=array.size();
    int col=array[0].size();


    int* hostInput = new int[row*col];
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            hostInput[i*col+j]=array[i][j];
        }
    }

    int* deviceInput;
    hipMalloc((void**)&deviceInput, row * col * sizeof(int));
    hipMemcpy(deviceInput, hostInput, row * col * sizeof(int), hipMemcpyHostToDevice);

    float* hostOutput = new float[row * col];
    float* deviceOutput;
    hipMalloc((void**)&deviceOutput, row * col * sizeof(float));

    // 定义CUDA的网格和块大小
    int gridSize = (row * col + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int blockSize = BLOCK_SIZE;

    // 预计算对数表
    double* logTable=new double[26];
    for (int k = 1; k <= 25; k++)
    {
        logTable[k] = log2(double(k));
        //printf("%f\n",logTable[k-1]);
    }


    calculateEntropy<<<gridSize, blockSize>>>(deviceInput, deviceOutput, row,col);
    hipMemcpy(hostOutput, deviceOutput, row* col * sizeof(float), hipMemcpyDeviceToHost);


    std::vector<std::vector<float> > entropyArray;
    entropyArray.resize(row);
    for(int i=0;i<row;i++){
        entropyArray[i].resize(col);
    }
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            array[i][j]=hostInput[i*col+j];
            entropyArray[i][j]=hostOutput[i*col+j];
        }
    }

    write(array,entropyArray,"output/cuda_bl_lu.in","output/cuda_bl_lu.out");
    

    // 释放内存
    delete[] hostInput;
    delete[] hostOutput;
    hipFree(deviceInput);
    hipFree(deviceOutput);


    return 0;
}
