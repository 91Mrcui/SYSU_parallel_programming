#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#define BLOCK_SIZE 16
#include "deal_binary.h"
using std::string;

// CUDA核函数，计算以每个元素为中心的窗口中的熵
__global__ void calculateEntropy(int* input, float* output, int row,int col)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < row * col)
    {
        int r = index / col;
        int c = index % col;

        int windowSize = 5;
        int windowStartRow = r - 2;
        int windowStartCol = c - 2;
        int windowEndRow = windowStartRow + 4;
        int windowEndCol = windowStartCol + 4;

        // 边界处理
        if (windowStartRow < 0)
            windowStartRow = 0;
        if (windowStartCol < 0)
            windowStartCol = 0;
        if (windowEndRow >= row)
            windowEndRow = row - 1;
        if (windowEndCol >= col)
            windowEndCol = col - 1;

        float entropy = 0.0f;
        int windowElements = (windowEndRow - windowStartRow + 1) * (windowEndCol - windowStartCol + 1);

        // 计算窗口内元素的频率
        int frequency[16] = { 0 };
        for (int i = windowStartRow; i <= windowEndRow; i++)
        {
            for (int j = windowStartCol; j <= windowEndCol; j++)
            {
                int value = input[i * col + j];
                frequency[value]++;
            }
        }

        // 计算熵
        for (int k = 0; k < 16; k++)
        {
            float prob = static_cast<float>(frequency[k]) / windowElements;
            if (prob > 0.0f)
                entropy -= prob * log2f(prob);
        }

        output[index] = entropy;
    }
}

int main()
{
    int sizes[] = { 5, 16, 128, 1024, 2048, 3000 ,4000};
    srand(static_cast<unsigned int>(1234));
    /*
    for (int i = 0; i < 7; i++)
    {
        int size = sizes[i];

        // 随机生成二维数组
        int* hostInput = new int[size * size];
        for (int j = 0; j < size * size; j++)
            hostInput[j] = rand() % 16;

        int* deviceInput;
        hipMalloc((void**)&deviceInput, size * size * sizeof(int));
        hipMemcpy(deviceInput, hostInput, size * size * sizeof(int), hipMemcpyHostToDevice);

        float* hostOutput = new float[size * size];
        float* deviceOutput;
        hipMalloc((void**)&deviceOutput, size * size * sizeof(float));

        // 定义CUDA的网格和块大小
        int gridSize = (size * size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        int blockSize = BLOCK_SIZE;
        

        clock_t start, finish;
    	//clock_t为CPU时钟计时单元数
    	start = clock();
        // 调用CUDA核函数
        calculateEntropy<<<gridSize, blockSize>>>(deviceInput, deviceOutput, size);

        hipMemcpy(hostOutput, deviceOutput, size * size * sizeof(float), hipMemcpyDeviceToHost);
        finish=clock();
        // 输出结果
        std::cout << "Array Size: " << size << " using time: "<<1000*double(finish - start) / CLOCKS_PER_SEC<<" ms"<<std::endl;
        //std::cout << "Array Size: " << size << std::endl;
        for (int row = 0; row < size; row++)
        {
            for (int col = 0; col < size; col++)
            {
                //std::cout << std::fixed << std::setprecision(5) << hostOutput[row * size + col] << " ";
            }
            //std::cout << std::endl;
        }

        // 释放内存
        delete[] hostInput;
        delete[] hostOutput;
        hipFree(deviceInput);
        hipFree(deviceOutput);
    }
    */
    std::vector<std::vector<int> > array;
    std::vector<float> res;
    read(array,res,"test/test.in","test/test.out");
    int row=array.size();
    int col=array[0].size();


    int* hostInput = new int[row*col];
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            hostInput[i*col+j]=array[i][j];
        }
    }

    int* deviceInput;
    hipMalloc((void**)&deviceInput, row * col * sizeof(int));
    hipMemcpy(deviceInput, hostInput, row * col * sizeof(int), hipMemcpyHostToDevice);

    float* hostOutput = new float[row * col];
    float* deviceOutput;
    hipMalloc((void**)&deviceOutput, row * col * sizeof(float));

    // 定义CUDA的网格和块大小
    int gridSize = (row * col + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int blockSize = BLOCK_SIZE;
    calculateEntropy<<<gridSize, blockSize>>>(deviceInput, deviceOutput, row,col);
    hipMemcpy(hostOutput, deviceOutput, row* col * sizeof(float), hipMemcpyDeviceToHost);


    std::vector<std::vector<float> > entropyArray;
    entropyArray.resize(row);
    for(int i=0;i<row;i++){
        entropyArray[i].resize(col);
    }
    for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            //array[i][j]=hostInput[i*col+j];
            entropyArray[i][j]=hostOutput[i*col+j];
        }
    }

    write(array,entropyArray,"output/cuda_bl.in","output/cuda_bl.out");
    

    // 释放内存
    delete[] hostInput;
    delete[] hostOutput;
    hipFree(deviceInput);
    hipFree(deviceOutput);
    return 0;
}
